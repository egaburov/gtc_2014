#include "hip/hip_runtime.h"
// Sean Baxter's GTC talk companion source.
// http://nvlabs.github.io/moderngpu/

#include "common.cuh"

// Search from needles A into haystack B. Returns lower- or upper-bound indices
// for all A needles.
template<int NT, int VT>
__global__ void KernelLBSSimple(int aCount, const int* b_global, int bCount,
	int* indices_global, SearchBounds bounds) {

	const int NV = NT * VT;
	__shared__ int data_shared[NT * VT + 1];

	int tid = threadIdx.x;

	// Load bCount elements from B.
	T x[VT];
	#pragma unroll
	for(int i = 0; i < VT; ++i) {
		int index = NT * i + tid;
		if(index < bCount) x[i] = b_global[index];
	}
	
	// Store all elements to shared memory.
	#pragma unroll
	for(int i = 0; i < VT; ++i)
		data_shared[NT * i + tid] = x[i];
	__syncthreads();


	// Each thread searches for its Merge Path partition.
	int diag = VT * tid;
	int begin = max(0, diag - bCount);
	int end = min(diag, aCount);

	while(begin < end) {
		int mid = (begin + end)>> 1;
		int aKey = mid;
		int bKey = data_shared[diag - 1 - mid];
		bool pred = !(aKey < bKey);
		if(pred) begin = mid + 1;
		else end = mid;
	}
	int mp = begin;


	// Sequentially merge into register starting from the partition.
	int a = mp;
	int b = aCount + diag - a;
	int aStart = a;

	int indices[VT];
	int decisions = 0;

	#pragma unroll
	for(int i = 0; i < VT; ++i) {
		bool p;
		if(b >= NV) p = true;
		else if(a >= aCount) p = false;
		else p = !(data_shared[b] < data_shared[a]);
		
		if(p) {
			// aKey is smaller than bKey. Save bKey's index as the result of 
			// the search and advance to the next needle A.
			indices[i] = b - aCount;
			decisions |= 1<< i;
			++a;
		} else {
			// bKey is smaller than aKey. Advance to the next b.
			++b;
		}
	}
	__syncthreads();

	// Compact the indices to shared memory.
	#pragma unroll
	for(int i = 0; i < VT; ++i)
		if((1<< i) & decisions)
			data_shared[aStart++] = indices[i];
	__syncthreads();

	// Store all aCount indices to global memory.
	for(int i = tid; i < aCount; i += NT)
		indices_global[i] = data_shared[i];
}


int main(int argc, char** argv) {

	const int NT = 128;
	const int VT = 7;
	const int NV = NT * VT;

	int aCount = NV / 7;
	int bCount = NV - aCountA;

	// Generate random sorted arrays to merge.
	std::vector<int> aHost(aCount), bHost(bCount);
	for(int i = 0; i < aCount; ++i)
		aHost[i] = rand() % 100;
	for(int i = 0; i < bCount; ++i)
		bHost[i] = rand() % 100;

	std::sort(aHost.begin(), aHost.end());
	std::sort(bHost.begin(), bHost.end());

	int* a_global, *b_global;
	cudaMalloc2(&a_global, aHost);
	cudaMalloc2(&b_global, bHost);

	int* indices_global;
	cudaMalloc2(&indices_global, aCount);

	KernelSortedSearchSimple<NT, VT><<<1, NT>>>(a_global, aCount, b_global, 
		bCount, indices_global, SearchBoundsLower);

	std::vector<int> indicesHost(aCount);
	copyDtoH(&indicesHost[0], indices_global, aCount);

	hipFree(a_global);
	hipFree(b_global);
	hipFree(indices_global);

	for(int a = 0; a < aCount; ++a) {
		printf("Key %3d  index %3d\n", aHost[a], indicesHost[a]);

		// Print all the keys behind it.
		int begin = indicesHost[a];
		int end = (a + 1 < aCount) ? indicesHost[a + 1] : bCount;
		int count = end - begin;

		for(int i = 0; i < count; ++i) {
			if(0 == (i % 5)) {
				if(i) printf("\n");
				printf("\t%3d: ", begin + i);
			}
			printf("%3d  ", bHost[begin + i]);
		}
		printf("\n");
	}
	return 0;
}